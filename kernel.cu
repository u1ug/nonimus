#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>

#include <stdio.h>
#include "kernel.h"


extern "C" {

    __global__ void addKernel(float A[10000][10000], float B[10000][10000], float C[10000][10000])
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        if(i < 1000 && j < 1000) C[i][j] = A[i][j] + B[i][j];
    }
        
    void addWithCuda(float A[10000][10000], float B[10000][10000], float C[10000][10000], unsigned int numBlocks, unsigned int threads)
    {
        dim3 blockSize;
        dim3 gridSize;
        int threadNum;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        threadNum = 1024;
        blockSize = dim3(threadNum, 1, 1);
        gridSize = dim3(numCols/threadNum+1, numRows, 1);
        addKernel << <numBlocks, threadsPerBlock >> > (A, B, C);
    }
}

// Helper function for using CUDA to add vectors in parallel.
