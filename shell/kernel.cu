#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>

#include <stdio.h>
#include "kernel.h"


extern "C" {

    __global__ void addKernel(float A[1000][1000], float B[1000][1000], float C[1000][1000])
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
            C[i][j] = A[i][j] + B[i][j];
    }
        
    void addWithCuda(float A[1000][1000], float B[1000][1000], float C[1000][1000], unsigned int numBlocks, unsigned int threads)
    {
        dim3 threadsPerBlock(1000, 1000);
        addKernel << <numBlocks, threadsPerBlock >> > (A, B, C);
    }
}

// Helper function for using CUDA to add vectors in parallel.
